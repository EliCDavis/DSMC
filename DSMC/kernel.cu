#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>

/* Personal declarations */
#include "vect3d.h"
#include "particle.h"


// Physical constant describing atom collision size
const float sigmak = 1e-28; // collision cross section

// Note, pnum recomputed from mean particle per cell and density
float pnum = 1e27; // number of particles per simulated particle


hipError_t initializeCuda(hiprandState_t* states, int blocks);

hipError_t inflowPotentialParticles(hiprandState_t* randomStates, particle* particleList, int i, int j, int k, int meanParticlePerCell);


int main()
{
	int numberOfInflowParticlesEachStep = 1 * 2 * 3 * 4;

	hiprandState_t* dev_randomInflowStates = NULL;
	
	hipError_t cudaStatus = initializeCuda(dev_randomInflowStates, numberOfInflowParticlesEachStep);

	particle *inflowParticleList = (particle*)malloc(numberOfInflowParticlesEachStep * sizeof(particle));

	inflowPotentialParticles(dev_randomInflowStates, inflowParticleList, 1, 2, 3, 4);
	
	for (int i = 0; i < numberOfInflowParticlesEachStep ; ++i)
	{
		printf(
			"[%-2d] vel{ %.3f, %.3f, %.3f }; pos{ %.3f, %.3f, %.3f };\n",
			i,
			inflowParticleList[i].velocity.x,
			inflowParticleList[i].velocity.y,
			inflowParticleList[i].velocity.z,
			inflowParticleList[i].position.x,
			inflowParticleList[i].position.y,
			inflowParticleList[i].position.z
		);
	}

	//hipFree(dev_randomInflowStates);

	printf("Complete");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

/* ============================== INITIALIZE =============================== */

__global__ void initRandomStates(unsigned int seed, hiprandState_t* states) {
	//int idx = blockIdx.x * blockDim.x + threadIdx.x;
	///* we have to initialize the state */
	//hiprand_init(0, /* the seed can be the same for each core, here we pass the time in from the CPU */
	//	idx, /* the sequence number should be different for each core (unless you want all
	//				cores to get the same sequence of numbers for some reason - use thread id! */
	//	0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
	//	&states[idx]);

	//hiprand(&states[idx]);

	hiprandState_t state;

	/* we have to initialize the state */
	hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
		0, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&state);

	/* hiprand works like rand - except that it takes a state as a parameter */
	hiprand(&state);
}

hipError_t initializeCuda(hiprandState_t *randomInflowStates, int blocks)
{
	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t hipError_t = hipSetDevice(0);

	hiprandState_t *dev_states;

	//hipMalloc((void**) &dev_states, blocks * sizeof(hiprandState_t));
	//initRandomStates <<<1, blocks>>>(2, dev_states);
	
	//printf("%d\n", time(0))

	hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		printf("init launch failed: %s\n", hipGetErrorString(hipError_t));
		return hipError_t;
	}

	// hipDeviceSynchronize();

	return hipError_t;
}

/* ================================ INFLOW ================================= */

__global__ void inflowKernel(particle *particles)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState_t state;
	hiprand_init(0, /* the seed controls the sequence of random values that are produced */
		idx, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&state);

	particles[idx].position.x = hiprand_uniform(&state);
	particles[idx].position.y = hiprand_uniform(&state);
	particles[idx].position.z = hiprand_uniform(&state);

	particles[idx].velocity.x = hiprand_uniform(&state);
	particles[idx].velocity.y = hiprand_uniform(&state);
	particles[idx].velocity.z = hiprand_uniform(&state);

	particles[idx].index = 1;
}

/* 
 Fill an array with new random particles to be exeucuted on

 Notes:
	Do I even need dev_a? Can I just use particle list?
 */
hipError_t inflowPotentialParticles(hiprandState_t *randomStates, particle *particleList, int i, int j, int k, int meanParticlePerCell) {
	int numOfPoints = i*j*k*meanParticlePerCell;
	int size = numOfPoints * sizeof(particle);
	particle *dev_a;

	hipError_t cudaStatus = hipMalloc((void**)&dev_a, size);
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}

	inflowKernel <<<1, numOfPoints>>>(dev_a);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("inflow launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}
	return hipMemcpy(particleList, dev_a, size, hipMemcpyDeviceToHost);
}
