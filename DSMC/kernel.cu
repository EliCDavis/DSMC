#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "vect3d.h"
#include "particle.h"
#include "cell.h"
#include "collisionInfo.h"


#include <iostream>
#include <fstream>

// 970 Constraints
#define MAX_THREAD_PER_BLOCK 1024
#define MAX_THREAD_PER_PROCESOR 2048
#define NUM_OF_BLOCKS 2048

#define HIP_PI_F 3.141592654f

// Geometry
#define PLATE_X -0.25
#define PLATE_DY 0.25
#define PLATE_DZ 0.5

// Physical constant describing atom collision size
const float sigmak = 1e-28; // collision cross section

// Note, pnum recomputed from mean particle per cell and density
float pnum = 1e27; // number of particles per simulated particle

using namespace std;


hipError_t initializeCuda(hiprandState_t** states, int blocks, cell** deviceCells, int numCells);

hipError_t inflowPotentialParticles(hiprandState_t* randomStates, particle* particleList, vect3d cellDimensions, int meanParticlePerCell, float vmean, float vtemp);

hipError_t moveAndIndexParticles(particle* particleList, int numOfParticles, float deltaTime, vect3d cellDimensions, vect3d dividedCellDimensions, int* numToDelete);

particle* removeParticlesOutofBounds(particle* particles, int size, int newSize);

hipError_t clearCellInformation(cell* cells, int numCells);

void sampleCellInformation(particle* particles, int numOfParticles, cell* cells, int numCells, int* cellSteal, bool* thingsHaveChanged);

__device__ void swapParticles(particle* p1, particle* p2) {
	int tempIndex = p1->index;
	bool tempHitParticle = p1->hitParticle;
	bool tempHitPlate = p1->hitPlate;
	bool tempDelete = p1->deleteMe;
	
	float tempVelX = p1->velocity.x;
	float tempVelY = p1->velocity.y;
	float tempVelZ = p1->velocity.z;
	
	float tempPosX = p1->position.x;
	float tempPosY = p1->position.y;
	float tempPosZ = p1->position.z;
	
	p1->index = p2->index;
	p1->hitParticle = p2->hitParticle;
	p1->hitPlate = p2->hitPlate;
	p1->deleteMe = p2->deleteMe;

	p1->velocity.x = p2->velocity.x;
	p1->velocity.y = p2->velocity.y;
	p1->velocity.z = p2->velocity.z;

	p1->position.x = p2->position.x;
	p1->position.y = p2->position.y;
	p1->position.z = p2->position.z;

	p2->index = tempIndex;
	p2->hitParticle = tempHitParticle;
	p2->hitPlate = tempHitPlate;
	p2->deleteMe = tempDelete;

	p2->velocity.x = tempVelX;
	p2->velocity.y = tempVelY;
	p2->velocity.z = tempVelZ;

	p2->position.x = tempPosX;
	p2->position.y = tempPosY;
	p2->position.z = tempPosZ;
}

__global__ void bitonic_sort_step(particle *dev_values, int j, int k, int numParticles)
{
	unsigned int i, ixj; /* Sorting partners: i and ixj */
	i = blockDim.x * blockIdx.x + threadIdx.x;
	ixj = i^j;
	
	if (i >= numParticles || ixj >= numParticles) {
		return;
	}

	/* The threads with the lowest ids sort the array. */
	if ((ixj)>i) {
		if (((i&k) == 0 && dev_values[i].index > dev_values[ixj].index) || (((i&k) != 0) && dev_values[i].index < dev_values[ixj].index)) {
			swapParticles(&dev_values[i], &dev_values[ixj]);
		}
	}
}

/**
* Inplace bitonic sort using CUDA.
*/
void bitonic_sort_particles(particle *values, int numParticles)
{
	hipError_t status;
	int numberOfBlocks = ceil(double(numParticles) / double(MAX_THREAD_PER_BLOCK));

	particle *dev_values;
	size_t size = numParticles * sizeof(particle);

	status = hipMalloc((void**)&dev_values, size);
	status = hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

	int j, k;
	/* Major step */
	for (k = 2; k <= numParticles; k <<= 1) {
		/* Minor step */
		for (j = k >> 1; j>0; j = j >> 1) {
			bitonic_sort_step <<<numberOfBlocks, MAX_THREAD_PER_BLOCK >>>(dev_values, j, k, numParticles);
			status = hipGetLastError();
		}
	}
	status = hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
	status = hipFree(dev_values);
}


void collideParticles(
	particle* particleList,
	int particleListSize,
	collisionInfo* collisionData,
	cell* cellData,
	int cellDataSize,
	int nsample,
	float cellvol,
	float deltaT,
	hiprandGenerator_t cudaRandomHostGenerator);

void printParticle(particle p)
{
	printf(
		"i %d; v{ %.3f, %.3f, %.3f }; p{ %.3f, %.3f, %.3f };\n",
		p.index,
		p.velocity.x,
		p.velocity.y,
		p.velocity.z,
		p.position.x,
		p.position.y,
		p.position.z
	);
}

float ranf() {
	return rand() / 32767.0f;
}

void initializeCollision(collisionInfo *collisionData, int size ,float vtemp)
{
	for (int i = 0; i < size; ++i)
	{
		collisionData[i].maxCollisionRate = sigmak * vtemp;
		collisionData[i].collisionRemainder = ranf();
	}
}


void writeParticles(int step, particle* particles, int num) {
	ofstream myfile;
	myfile.open("output.txt");
	for (int p = 0; p < num; p++) {
		int status = particles[p].hitPlate;
		myfile << particles[p].position.x << " " << particles[p].position.y << " " << particles[p].position.z << " " << status << endl;
	}
	myfile.close();
}

int main()
{
	int meanParticlePerCell = 10;
	vect3d cellDimensions = vect3d(32, 32, 32);
	vect3d dividedCellDimensions = vect3d(2. / cellDimensions.x, 2. / cellDimensions.y, 2. / cellDimensions.z);
	float vmean = 1;
	float Mach = 20;
	float vtemp = vmean / Mach;
	float deltax = 2. / float(fmax(fmax(cellDimensions.x, cellDimensions.y), cellDimensions.z));
	float deltaT = .1 * deltax / (vmean + vtemp);
	float density = 1e30; // Number of molecules per unit cube of space

	// Initialize cuda random generator so host can create random numbers
	hiprandGenerator_t cudaRandomHostGenerator;
	hiprandCreateGenerator(&cudaRandomHostGenerator, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(cudaRandomHostGenerator, 1234ULL);

	// simulate for 4 free-stream flow-through times
	float time = 8. / (vmean + vtemp);
	int numberOfTimesteps = 1 << int(ceil(log(time / deltaT) / log(2.0)));
	printf("Time: %.2f; Steps: %d\n", time, numberOfTimesteps);


	// re-sample 4 times during simulation
	const int sample_reset = numberOfTimesteps / 4;
	int nsample = 0;

	int numberOfInflowParticlesEachStep = cellDimensions.y * cellDimensions.z * meanParticlePerCell;

	int currentNumberOfParticles = 0;

	const int numberOfCells = cellDimensions.x * cellDimensions.y * cellDimensions.z;
	pnum = density * numberOfCells / float(meanParticlePerCell);

	cell* deviceCellSamples;
	

	collisionInfo* collisionData = (collisionInfo*)malloc(numberOfCells * sizeof(collisionInfo));
	initializeCollision(collisionData, numberOfCells, vtemp);

	hiprandState_t* deviceRandomInflowStates = NULL;
	
	hipError_t cudaStatus = initializeCuda(&deviceRandomInflowStates, numberOfInflowParticlesEachStep, &deviceCellSamples, numberOfCells);
	if (cudaStatus != hipSuccess) {
		printf("init kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	particle *allParticles = (particle*)malloc(numberOfInflowParticlesEachStep * sizeof(particle));
	particle *inflowParticleList = (particle*)malloc(numberOfInflowParticlesEachStep * sizeof(particle));

	clock_t totalTime;

	clock_t clockTime;

	int* deviceCellSteals;
	cudaStatus = hipMalloc((void**)&deviceCellSteals, numberOfCells * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("sample malloc failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	bool* deviceContentsChanged;
	cudaStatus = hipMalloc((void**)&deviceContentsChanged, sizeof(bool));
	if (cudaStatus != hipSuccess) {
		printf("sample malloc failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}
	totalTime = clock();
	for (int t = 0; t < numberOfTimesteps; t++) {

		clockTime = clock();

		cudaStatus = inflowPotentialParticles(deviceRandomInflowStates, inflowParticleList, cellDimensions, meanParticlePerCell, vmean, vtemp);
		if (cudaStatus != hipSuccess) {
			printf("inflow failed: %s\n", hipGetErrorString(cudaStatus));
			return 1;
		}
		// Combine new particles with existing
		particle* newTotal = (particle*)malloc((numberOfInflowParticlesEachStep + currentNumberOfParticles) * sizeof(particle));
		memcpy(newTotal, inflowParticleList, numberOfInflowParticlesEachStep * sizeof(particle));
		if (currentNumberOfParticles != 0) {
			memcpy(newTotal + numberOfInflowParticlesEachStep, allParticles, currentNumberOfParticles * sizeof(particle));
		}
		free(allParticles);
		allParticles = newTotal;
		currentNumberOfParticles += numberOfInflowParticlesEachStep;

		int numToDelete = 0;
		moveAndIndexParticles(allParticles, currentNumberOfParticles, deltaT, cellDimensions, dividedCellDimensions, &numToDelete);

		// Clean up list of particles out of bounds and recompute cell map
		particle* cleanedParticles = removeParticlesOutofBounds(allParticles, currentNumberOfParticles, currentNumberOfParticles - numToDelete);
		currentNumberOfParticles -= numToDelete;
		free(allParticles);
		allParticles = cleanedParticles;

		if (t % sample_reset == 0)
		{
			clearCellInformation(deviceCellSamples, numberOfCells);
			nsample = 0;
		}
		nsample++;
		
		if (t % 37 == 0) {
			bitonic_sort_particles(allParticles, currentNumberOfParticles);
		}

		sampleCellInformation(allParticles, currentNumberOfParticles, deviceCellSamples, numberOfCells, deviceCellSteals, deviceContentsChanged);

		collideParticles(allParticles,
			currentNumberOfParticles,
			collisionData,
			deviceCellSamples,
			numberOfCells,
			nsample,
			numberOfCells,
			deltaT,
			cudaRandomHostGenerator);

		clockTime = clock() - clockTime;
		printf("%d %f %d\n", t, ((double)clockTime) / CLOCKS_PER_SEC, currentNumberOfParticles);
	}

	printf("Total Time: %f", ((double)clock() - totalTime) / CLOCKS_PER_SEC);

	writeParticles(0, allParticles, currentNumberOfParticles);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	printf("Complete");

    return 0;
}

/* ============================== INITIALIZE =============================== */

__global__ void initRandomStatesKernel(unsigned int seed, hiprandState_t* states) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	/* we have to initialize the state */
	hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
		idx, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&states[idx]);
}

__global__ void initCellsKenel(cell* cells, int numCells) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= numCells) {
		return;
	}
	cells[idx].currentNumberOfParticles = 0;
	cells[idx].energy= 0;
	cells[idx].numberOfParticles = 0;
	cells[idx].velocity.x = 0;
	cells[idx].velocity.y = 0;
	cells[idx].velocity.z = 0;
}

hipError_t initializeCuda(hiprandState_t **randomInflowStates, int numOfStates, cell** deviceCells, int numCells)
{
	int blockSize = numOfStates / NUM_OF_BLOCKS;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t hipError_t = hipSetDevice(0);

	hiprandState_t *dev_states;

	hipError_t = hipMalloc((void**) &dev_states, numOfStates * sizeof(hiprandState_t));
	if (hipError_t != hipSuccess) {
		printf("init malloc failed: %s\n", hipGetErrorString(hipError_t));
		return hipError_t;
	}

	initRandomStatesKernel <<<NUM_OF_BLOCKS, blockSize >>>(2, dev_states);
	hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		printf("init kernel launch failed: %s\n", hipGetErrorString(hipError_t));
		return hipError_t;
	}

	*randomInflowStates = dev_states;

	cell* devcells;

	hipError_t = hipMalloc((void**)&devcells, numCells * sizeof(cell));
	if (hipError_t != hipSuccess) {
		printf("init malloc failed: %s\n", hipGetErrorString(hipError_t));
		return hipError_t;
	}

	int numberOfBlocks = ceil(double(numCells) / double(MAX_THREAD_PER_BLOCK));

	initCellsKenel<<<numberOfBlocks, MAX_THREAD_PER_BLOCK >>>(devcells, numCells);

	*deviceCells = devcells;

	return hipGetLastError();
}

/* ================================ HELPERS ================================ */

/*
	Compute a unit vector with a random orientation and uniform distribution
*/
__device__ void randomDirection(hiprandState_t seed, vect3d* vel)
{
	vel[0].x = 2.0 * hiprand_uniform(&seed) - 1;
	double A = sqrt(1. - vel[0].x * vel[0].x);
	double theta = hiprand_uniform(&seed) * 2 * HIP_PI_F;
	vel[0].y = A * cos(theta);
	vel[0].z = A * sin(theta);
}

/* =============================== 1. INFLOW =============================== */

__global__ void inflowKernel(int numOfParticles, hiprandState_t *randState, particle *particles, int dimX, int dimY, int dimZ, float vmean, float vtemp)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= numOfParticles) {
		return;
	}

	int k = idx % (dimX* dimY);
	int cellY = k % int(dimY);
	int cellZ = int(floorf(float(k) / float(dimY)));

	double dx = 2. / float(dimX);
	double dy = 2. / float(dimY);
	double dz = 2. / float(dimZ);

	double cx = -1 - dx;
	double cy = -1 + float(cellY) * dy;
	double cz = -1 + float(cellZ) * dz;

	particles[idx].position.x = cx + hiprand_uniform(&randState[idx]) * dx;
	particles[idx].position.y = cy + hiprand_uniform(&randState[idx]) * dy;
	particles[idx].position.z = cz + hiprand_uniform(&randState[idx]) * dz;

	randomDirection(randState[idx], &particles[idx].velocity);

	double rndVel = sqrt(-log(fmax(double(sqrt(hiprand_uniform(&randState[idx]))), 1e-200))) * vtemp;

	particles[idx].velocity.x = (particles[idx].velocity.x * rndVel) + vmean;
	particles[idx].velocity.y = particles[idx].velocity.y * rndVel;
	particles[idx].velocity.z = particles[idx].velocity.z * rndVel;

	particles[idx].index = 1;
	particles[idx].deleteMe = false;
	particles[idx].hitPlate = false;
	particles[idx].hitParticle= false;
}

/* 
 Fill an array with new random particles to be exeucuted on
 */
hipError_t inflowPotentialParticles(hiprandState_t* deviceRandomStates, particle *particleList, vect3d cellDimensions, int meanParticlePerCell, float vmean, float vtemp) {
	
	int numOfPoints = cellDimensions.y * cellDimensions.z * meanParticlePerCell;

	int numberOfBlocks = ceil(double(numOfPoints) / double(MAX_THREAD_PER_BLOCK));

	int size = numOfPoints * sizeof(particle);

	particle *dev_a;
	hipError_t cudaStatus = hipMalloc((void**)&dev_a, size);
	if (cudaStatus != hipSuccess) {
		printf("inflow malloc failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	inflowKernel <<<numberOfBlocks, MAX_THREAD_PER_BLOCK >>>(numOfPoints, deviceRandomStates, dev_a, cellDimensions.x, cellDimensions.y, cellDimensions.z, vmean, vtemp);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("inflow launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(particleList, dev_a, size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("inflow memcpy failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	return hipFree(dev_a);
}



/* =========================== 2. MOVE PARTICLES =========================== */

__global__ void moveParticlesKernel(int* deviceDeletionCount, particle* particles, int numParticles, float deltaTime, int dimX, int dimY, int dimZ, float divX, float divY, float divZ) {
	
	extern __shared__ int sdata[];
	sdata[threadIdx.x] = 0;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= numParticles) {
		return;
	}

	float newPosX = particles[idx].position.x + (particles[idx].velocity.x * deltaTime);
	float newPosY = particles[idx].position.y + (particles[idx].velocity.y * deltaTime);
	float newPosZ = particles[idx].position.z + (particles[idx].velocity.z * deltaTime);


	// Where did it actually pass through the plate..
	double t = (particles[idx].position.x - PLATE_X) / (particles[idx].position.x - newPosX);
	float pointOfCollisionY = (particles[idx].position.y * (1.0 - t)) + (newPosY * t);
	float pointOfCollisionZ = (particles[idx].position.z * (1.0 - t)) + (newPosZ * t);

	// Actually collided..
	if (((particles[idx].position.x < PLATE_X && newPosX > PLATE_X) ||
		(particles[idx].position.x > PLATE_X && newPosX < PLATE_X)) && (pointOfCollisionY < PLATE_DY && pointOfCollisionY > -PLATE_DY) &&
		(pointOfCollisionZ < PLATE_DZ && pointOfCollisionZ > -PLATE_DZ))
	{
		newPosX = newPosX - 2 * (newPosX - PLATE_X);
		particles[idx].velocity.x = -particles[idx].velocity.x;
		particles[idx].hitPlate = true;
	}


	newPosY = newPosY + (2.0 * ((newPosY < -1) - (newPosY > 1)));
	newPosZ = newPosZ + (2.0 * ((newPosZ < -1) - (newPosZ > 1)));

	// Assign particle positions and index
	particles[idx].position.x = newPosX;
	particles[idx].position.y = newPosY;
	particles[idx].position.z = newPosZ;

	int i = int(fmin(floor((newPosX + 1.0) / divX), double(dimX - 1)));
	int j = int(fmin(floor((newPosY + 1.0) / divY), double(dimY - 1)));
	int k = int(fmin(floor((newPosZ + 1.0) / divZ), double(dimZ - 1)));
	particles[idx].index = i * dimY * dimZ + j * dimZ + k;

	int deletion = 0;
	if (newPosX > 1.0 || newPosX < -1.0) {
		deletion = 1;
		particles[idx].deleteMe = true;
	}

	sdata[threadIdx.x] = deletion;
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
		if (threadIdx.x < s) {
			sdata[threadIdx.x] += sdata[threadIdx.x + s];
		}
		__syncthreads();
	}
	if (threadIdx.x == 0) {
		deviceDeletionCount[blockIdx.x] = sdata[0];
	}
}

/*
	Move particles appropriately and marks those out of bounds with a flag for deletion.
	Reindexes particles not marked for deletion
*/
hipError_t moveAndIndexParticles(particle* particleList, int numOfParticles, float deltaTime, vect3d cellDimensions, vect3d dividedCellDimensions, int* numToDelete) {
	int numberOfBlocks = ceil(double(numOfParticles) / double(MAX_THREAD_PER_BLOCK));
	
	particle *dev_a;

	int *deviceDeletionCount;

	hipError_t cudaStatus = hipMalloc((void**)&dev_a, numOfParticles*sizeof(particle));
	if (cudaStatus != hipSuccess) {
		printf("move&index malloc failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&deviceDeletionCount, numberOfBlocks * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("move&index malloc failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(dev_a, particleList, numOfParticles * sizeof(particle), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("move&index memcpy to device failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	moveParticlesKernel <<<numberOfBlocks, MAX_THREAD_PER_BLOCK, MAX_THREAD_PER_BLOCK * sizeof(int) >>>(deviceDeletionCount, dev_a, numOfParticles, deltaTime, cellDimensions.x, cellDimensions.y, cellDimensions.z, dividedCellDimensions.x, dividedCellDimensions.y, dividedCellDimensions.z);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("move&index launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}


	cudaStatus = hipMemcpy(particleList, dev_a, numOfParticles * sizeof(particle), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("move&index memcpy to host failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	int* deletionCount = (int*)malloc(numberOfBlocks * sizeof(int));
	cudaStatus = hipMemcpy(deletionCount, deviceDeletionCount, numberOfBlocks * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("move&index memcpy to host failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	for (int i = 0; i < numberOfBlocks; i++) {
		*numToDelete += deletionCount[i];
	}
	free(deletionCount);

	cudaStatus = hipFree(deviceDeletionCount);
	if (cudaStatus != hipSuccess) {
		printf("move&index free count failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	return hipFree(dev_a);
}

/* ========================== 3. REMOVE PARTICLES ========================== */

particle* removeParticlesOutofBounds(particle* particles, int originalParticleSize, int newSize) {
	particle* newParticleList = (particle*)malloc(newSize * sizeof(particle));
	int added = 0;
	for (int i = 0; i < originalParticleSize; i++) {
		if (!particles[i].deleteMe) {
			newParticleList[added] = particle(particles[i]);
			added += 1;
		}
	}
	return newParticleList;
}

/* ============================== 4. SAMPLING ============================== */

__global__ void clearCellsKernel(cell* cells, int numCells) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= numCells) {
		return;
	}
	cells[idx].numberOfParticles = 0;
	cells[idx].velocity.x = 0;
	cells[idx].velocity.y = 0;
	cells[idx].velocity.z = 0;
	cells[idx].energy = 0;
}

hipError_t	clearCellInformation(cell* deviceCells, int numCells) {
	int numberOfBlocks = ceil(double(numCells) / double(MAX_THREAD_PER_BLOCK));

	clearCellsKernel <<<numberOfBlocks, MAX_THREAD_PER_BLOCK >>>(deviceCells, numCells);

	return hipGetLastError();
}

__global__ void initializeStolenKernel(bool* stolenBefore, int numStolen) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= numStolen) {
		return;
	}
	stolenBefore[idx] = false;
}

__global__ void sameplCellClearKernel(cell* cells, int numCells) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= numCells) {
		return;
	}
	cells[idx].currentNumberOfParticles = 0;
}

__global__ void sameplCellStealKernel(particle* particles, int particleSize, int* cellSteals, bool* stolenBefore, bool* changed) {
	*changed = false;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= particleSize || stolenBefore[idx]) {
		return;
	}
	cellSteals[particles[idx].index] = idx;
}

__global__ void sameplCellRunKernel(particle* particles, int particleSize, int* cellSteals, cell* cells, bool* stolenBefore, bool* changed) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= particleSize) {
		return;
	}
	int cellIndex = particles[idx].index;

	// We didn't succesfully grab the cell, try again next time
	if (cellSteals[cellIndex] != idx) {
		return;
	}

	cells[cellIndex].currentNumberOfParticles++;
	cells[cellIndex].numberOfParticles++;
	cells[cellIndex].velocity.x += particles[idx].velocity.x;
	cells[cellIndex].velocity.y += particles[idx].velocity.y;
	cells[cellIndex].velocity.z += particles[idx].velocity.z;
	cells[cellIndex].energy  += (.5 *
		((particles[idx].velocity.x * particles[idx].velocity.x) +
		 (particles[idx].velocity.y * particles[idx].velocity.y) +
		 (particles[idx].velocity.z * particles[idx].velocity.z)));

	cellSteals[cellIndex] = -1;
	stolenBefore[idx] = true;
	*changed = true;
}


void sampleCellInformation(particle* particles, int numOfParticles, cell* deviceCells, int numCells, int* deviceCellsStolen, bool* deviceThingsChanged) {

	int numberOfBlocksForParticles = ceil(double(numOfParticles) / double(MAX_THREAD_PER_BLOCK));
	int numberOfBlocksForCells = ceil(double(numCells) / double(MAX_THREAD_PER_BLOCK));

	particle *deviceParticles;
	hipError_t cudaStatus = hipMalloc((void**)&deviceParticles, numOfParticles * sizeof(particle));
	if (cudaStatus != hipSuccess) {
		printf("sample malloc failed: %s\n", hipGetErrorString(cudaStatus));
		return ;
	}

	bool* deviceParticleSuccessStolen;
	cudaStatus = hipMalloc((void**)&deviceParticleSuccessStolen, numOfParticles * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		printf("sample malloc failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	initializeStolenKernel <<<numberOfBlocksForParticles, MAX_THREAD_PER_BLOCK>>> (deviceParticleSuccessStolen, numOfParticles);

	cudaStatus = hipMemcpy(deviceParticles, particles, numOfParticles * sizeof(particle), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("sample memcpy to device failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	sameplCellClearKernel <<<numberOfBlocksForCells, MAX_THREAD_PER_BLOCK>>> (deviceCells, numCells);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("sample kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	bool thingsChanged = true;
	int times = 0;
	while (thingsChanged) {
		sameplCellStealKernel <<<numberOfBlocksForParticles, MAX_THREAD_PER_BLOCK>>>(deviceParticles, numOfParticles, deviceCellsStolen, deviceParticleSuccessStolen, deviceThingsChanged);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			printf("sample kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return;
		}

		sameplCellRunKernel <<<numberOfBlocksForParticles, MAX_THREAD_PER_BLOCK>>>(deviceParticles, numOfParticles, deviceCellsStolen, deviceCells, deviceParticleSuccessStolen, deviceThingsChanged);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			printf("sample kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return;
		}

		if (times >= 3) {
			cudaStatus = hipMemcpy(&thingsChanged, deviceThingsChanged, sizeof(bool), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				printf("sample copy failed: %s\n", hipGetErrorString(cudaStatus));
				return;
			}
			times = 0;
		}
		times++;
	}

	cudaStatus = hipFree(deviceParticles);
	if (cudaStatus != hipSuccess) {
		printf("sample free failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	cudaStatus = hipFree(deviceParticleSuccessStolen);
	if (cudaStatus != hipSuccess) {
		printf("sample free failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

}


/* ============================ Cell Collision ============================= */

__global__ void preCollisionKernel(cell* cells, collisionInfo* collisionData, int cellSize, float nSample, float deltaT, float cellVol, float moleculesPerParticle, int *numOfRandomNumbers) {

	extern __shared__ int sdata[];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[threadIdx.x] = 0;

	if (idx >= cellSize) {
		return;
	}

	// Compute mean and instantaneous particle numbers for the cell
	float n_mean = float(cells[idx].numberOfParticles) / float(nSample);
	float n_instant = cells[idx].currentNumberOfParticles;

	// Compute a number of particles that need to be selected for collision tests
	float select = n_instant * n_mean * moleculesPerParticle * collisionData[idx].maxCollisionRate * deltaT / cellVol + collisionData[idx].collisionRemainder;
	
	// We can only check an integer number of collisions in any timestep
	collisionData[idx].nSelect = int(select);

	// The remainder collision fraction is saved for next timestep
	collisionData[idx].collisionRemainder = select - float(collisionData[idx].nSelect);

	sdata[threadIdx.x] = collisionData[idx].nSelect * (n_instant >= 2) * 5;
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
		if (threadIdx.x < s) {
			sdata[threadIdx.x] += sdata[threadIdx.x + s];
		}
		__syncthreads();
	}
	if (threadIdx.x == 0) {
		numOfRandomNumbers[blockIdx.x] = sdata[0];
	}

}


// Computes a unit vector with a random orientation and uniform distribution
inline vect3d randomDir(float rndOne, float rndTwo)
{
	double B = 2. * rndOne - 1;
	double A = sqrt(1. - B * B);
	double theta = rndTwo * 2 * HIP_PI_F;
	return vect3d(B, A * cos(theta), A * sin(theta));
}

void collideParticles(
	particle* particleList,
	int particleListSize,
	collisionInfo* collisionData,
	cell* deviceCellData,
	int cellDataSize,
	int nsample, 
	float cellvol, 
	float deltaT,
	hiprandGenerator_t cudaRandomHostGenerator)
{

	// Compute number of particles per cell and compute a set of pointers
	// from each cell to the corresponding particles
	int* np = (int*)malloc(cellDataSize * sizeof(int));
	int* cnt = (int*)malloc(cellDataSize * sizeof(int));
	for (int i = 0; i < cellDataSize; ++i)
	{
		np[i] = 0;
		cnt[i] = 0;
	}

	for (int p = 0; p < particleListSize; p++)
	{
		np[particleList[p].index]++;
	}

	// Offsets will contain the index in the pmap data structure where
	// the pointers to particles for the given cell will begin
	int* offsets = (int*)malloc((cellDataSize + 1) * sizeof(int));
	offsets[0] = 0;
	for (int i = 0; i < cellDataSize; ++i)
	{
		offsets[i + 1] = offsets[i] + np[i];
	}

	// pmap is a structure of pointers from cells to particles, note
	// since there may be many particles per cell, the offsets need to
	// be used to access particles from this data structure.
	particle** pmap = (particle**)malloc(offsets[cellDataSize] * sizeof(particle*));
	for (int p = 0; p < particleListSize; p++)
	{
		int i = particleList[p].index;
		pmap[cnt[i] + offsets[i]] = &(particleList[p]);
		cnt[i]++;
	}

	free(cnt);
	
	collisionInfo* deviceCollisionData;
	hipError_t cudaStatus = hipMalloc((void**)&deviceCollisionData, cellDataSize * sizeof(collisionInfo));
	if (cudaStatus != hipSuccess) {
		printf("Pcol malloc failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	int* deviceRandomNumbersNeeded;
	cudaStatus = hipMalloc((void**)&deviceRandomNumbersNeeded, cellDataSize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("Pcol malloc failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	cudaStatus = hipMemcpy(deviceCollisionData, collisionData, cellDataSize * sizeof(collisionInfo), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("Pcol memcpy to device failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	int numberOfBlocks = ceil(double(cellDataSize) / double(MAX_THREAD_PER_BLOCK));

	preCollisionKernel<<<numberOfBlocks, MAX_THREAD_PER_BLOCK, MAX_THREAD_PER_BLOCK * sizeof(int)>>>(deviceCellData, deviceCollisionData, cellDataSize, nsample, deltaT, cellvol, pnum, deviceRandomNumbersNeeded);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("Pcol launch failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	int* blockAggRandomNumbersNeeded = (int*)malloc(numberOfBlocks * sizeof(int));
	cudaStatus = hipMemcpy(blockAggRandomNumbersNeeded, deviceRandomNumbersNeeded, numberOfBlocks * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("Pcol memcpy to host failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	int randomNumbersNeeded = 0;
	for (int i = 0; i < numberOfBlocks; i++) {
		randomNumbersNeeded += blockAggRandomNumbersNeeded[i];
	}
	free(blockAggRandomNumbersNeeded);

	float* randomNumbers = (float *)malloc(randomNumbersNeeded * sizeof(float));
	float* deviceRandomNumbers;
	cudaStatus = hipMalloc((void **)&deviceRandomNumbers, randomNumbersNeeded * sizeof(float));
	
	hiprandGenerateUniform(cudaRandomHostGenerator, deviceRandomNumbers, randomNumbersNeeded);
	cudaStatus = hipMemcpy(randomNumbers, deviceRandomNumbers, randomNumbersNeeded * sizeof(float), hipMemcpyDeviceToHost);
	cudaStatus = hipFree(deviceRandomNumbers);
	int randomNumbersUsed = 0;

	cudaStatus = hipMemcpy(collisionData, deviceCollisionData, cellDataSize * sizeof(collisionInfo), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("Pcol memcpy to host failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	// Loop over cells and select particles to perform collisions
	for (int cellIndex = 0; cellIndex < cellDataSize; ++cellIndex)
	{
		if (collisionData[cellIndex].nSelect > 0)
		{ // selected particles for collision
			if (np[cellIndex] < 2)
			{ // if not enough particles for collision, wait until
			  // we have enough
				collisionData[cellIndex].collisionRemainder += collisionData[cellIndex].nSelect;
			}
			else
			{
				// Select nselect particles for possible collision
				float cmax = collisionData[cellIndex].maxCollisionRate;
				for (int c = 0; c < collisionData[cellIndex].nSelect; ++c)
				{
					// select two points in the cell
					int pt1 = min(int(floor(randomNumbers[randomNumbersUsed] * np[cellIndex])), np[cellIndex] - 1);
					int pt2 = min(int(floor(randomNumbers[randomNumbersUsed + 1] * np[cellIndex])), np[cellIndex]- 1);

					// Make sure they are unique points
					while (pt1 == pt2) {
						pt2 = min(int(floor(ranf() * np[cellIndex])), np[cellIndex]- 1);
					}
					// Compute the relative velocity of two particles
					vect3d v1 = pmap[offsets[cellIndex] + pt1]->velocity;
					vect3d v2 = pmap[offsets[cellIndex] + pt2]->velocity;
					vect3d vr = v1 - v2;
					float vrm = norm(vr);

					// Compute collision  rate for hard sphere model
					float crate = sigmak * vrm;
					if (crate > cmax) {
						cmax = crate;
					}
					
					// Check if these particles actually collide
					if (randomNumbers[randomNumbersUsed + 2] < crate / collisionData[cellIndex].maxCollisionRate)
					{
						// Collision Accepted, adjust particle velocities
						// Compute center of mass velocity, vcm
						vect3d vcm = .5 * (v1 + v2);
						// Compute random perturbation that conserves momentum
						vect3d vp = randomDir(randomNumbers[randomNumbersUsed + 3], randomNumbers[randomNumbersUsed + 4]) * vrm;

						// Adjust particle velocities to reflect collision
						pmap[offsets[cellIndex] + pt1]->velocity = vcm + 0.5 * vp;
						pmap[offsets[cellIndex] + pt2]->velocity = vcm - 0.5 * vp;

						pmap[offsets[cellIndex] + pt1]->hitParticle = true;
						pmap[offsets[cellIndex] + pt2]->hitParticle = true;
					}
					randomNumbersUsed += 5;
				}
				// Update the maximum collision rate to be used in future timesteps
				// for determining number of particles to select.
				collisionData[cellIndex].maxCollisionRate = cmax;
			}
		}
	
	}

	free(pmap);
	free(np);
	free(offsets);
	free(randomNumbers);

	cudaStatus = hipFree(deviceCollisionData);
	if (cudaStatus != hipSuccess) {
		printf("Pcol free cell failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

}