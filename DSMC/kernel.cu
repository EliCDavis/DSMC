#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

/* Personal declarations */
#include "vect3d.h"
#include "particle.h"

#include <stdio.h>

hipError_t inflowPotentialParticles(particle* particleList, int i, int j, int k, int meanParticlePerCell);

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

// Physical constant describing atom collision size
const float sigmak = 1e-28; // collision cross section

// Note, pnum recomputed from mean particle per cell and density
float pnum = 1e27; // number of particles per simulated particle

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void inflowKernel(particle *particles)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	/*particles[idx] = particle(
		vect3d(blockIdx.x, blockDim.x, threadIdx.x), 
		vect3d(blockIdx.x, blockDim.x, threadIdx.x)
	);*/
	particles[idx].position.x = blockIdx.x;
	particles[idx].position.y = blockDim.x;
	particles[idx].position.z = threadIdx.x;

	particles[idx].velocity.x = blockIdx.x;
	particles[idx].velocity.y = blockDim.x;
	particles[idx].velocity.z = threadIdx.x;

	particles[idx].index = 1;

}

hipError_t initializeCuda()
{
	// Choose which GPU to run on, change this on a multi-GPU system.
	return hipSetDevice(0);
}

int main()
{
	/*
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

	*/

	hipError_t cudaStatus = initializeCuda();

	particle *particleList = 0;
	inflowPotentialParticles(particleList, 1, 2, 3, 4);

	printf("I think I completed?");
	printf("Maybe?");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

/* 
 Fill an array with new random particles to be exeucuted on
 */
hipError_t inflowPotentialParticles(particle *particleList, int i, int j, int k, int meanParticlePerCell) {
	
	int numOfPoints = i*j*k*meanParticlePerCell;
	int size = numOfPoints * sizeof(particle);
	particle *dev_a;
	particle *host_a = (particle*)malloc(size);

	hipError_t cudaStatus = hipMalloc((void**)&dev_a, size);
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}

	// Not sure if I have to do this, kinda just want them null right...
	// Copy empty particle list..
	/*cudaStatus = hipMemcpy(dev_a, a, size * sizeof(particle), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}*/

	inflowKernel <<<1, numOfPoints>>>(dev_a);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(host_a, dev_a, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < numOfPoints; ++i)
	{
		printf(
			"[%d] vel{ %f, %f, %f }\n", 
			i, 
			host_a[i].velocity.x, 
			host_a[i].velocity.y,
			host_a[i].velocity.z
		);
	}

	return cudaStatus;

}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
