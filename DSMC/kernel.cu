#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "vect3d.h"
#include "particle.h"
#include "cell.h"
#include "collisionInfo.h"


#include <iostream>
#include <fstream>

// 970 Constraints
#define MAX_THREAD_PER_BLOCK 1024
#define MAX_THREAD_PER_PROCESOR 2048
#define NUM_OF_BLOCKS 2048

#define HIP_PI_F 3.141592654f

// Geometry
#define PLATE_X -0.25
#define PLATE_DY 0.25
#define PLATE_DZ 0.5

// Physical constant describing atom collision size
const float sigmak = 1e-28; // collision cross section

// Note, pnum recomputed from mean particle per cell and density
float pnum = 1e27; // number of particles per simulated particle

using namespace std;


hipError_t initializeCuda(hiprandState_t** states, int blocks);

hipError_t inflowPotentialParticles(hiprandState_t* randomStates, particle* particleList, vect3d cellDimensions, int meanParticlePerCell, float vmean, float vtemp);

hipError_t moveAndIndexParticles(particle* particleList, int numOfParticles, float deltaTime, vect3d cellDimensions, vect3d dividedCellDimensions, int* numToDelete);

particle* removeParticlesOutofBounds(particle* particles, int size, int newSize);

hipError_t clearCellInformation(cell* cells, int numCells);

void sampleCellInformation(particle* particles, int numOfParticles, cell* cells, int numCells);

void collideParticles(
	particle* particleList,
	int particleListSize,
	collisionInfo* collisionData,
	cell* cellData,
	int cellDataSize,
	int nsample,
	float cellvol,
	float deltaT,
	hiprandGenerator_t cudaRandomHostGenerator);

void printParticle(particle p)
{
	printf(
		"i %d; v{ %.3f, %.3f, %.3f }; p{ %.3f, %.3f, %.3f };\n",
		p.index,
		p.velocity.x,
		p.velocity.y,
		p.velocity.z,
		p.position.x,
		p.position.y,
		p.position.z
	);
}

float ranf() {
	return rand() / 32767.0f;
}

void initializeCollision(collisionInfo *collisionData, int size ,float vtemp)
{
	for (int i = 0; i < size; ++i)
	{
		collisionData[i].maxCollisionRate = sigmak * vtemp;
		collisionData[i].collisionRemainder = ranf();
	}
}


void writeParticles(int step, particle* particles, int num) {
	ofstream myfile;
	myfile.open("output.txt");
	for (int p = 0; p < num; p++) {
		myfile << particles[p].position.x << " " << particles[p].position.y << " " << particles[p].position.z << " " << particles[p].status << endl;
	}
	myfile.close();
}

int main()
{
	int meanParticlePerCell = 10;
	vect3d cellDimensions = vect3d(32, 32, 32);
	vect3d dividedCellDimensions = vect3d(2. / cellDimensions.x, 2. / cellDimensions.y, 2. / cellDimensions.z);
	float vmean = 1;
	float Mach = 20;
	float vtemp = vmean / Mach;
	float deltax = 2. / float(fmax(fmax(cellDimensions.x, cellDimensions.y), cellDimensions.z));
	float deltaT = .1 * deltax / (vmean + vtemp);
	float density = 1e30; // Number of molecules per unit cube of space

	// Initialize cuda random generator so host can create random numbers
	hiprandGenerator_t cudaRandomHostGenerator;
	hiprandCreateGenerator(&cudaRandomHostGenerator, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(cudaRandomHostGenerator, 1234ULL);

	// simulate for 4 free-stream flow-through times
	float time = 8. / (vmean + vtemp);
	int numberOfTimesteps = 1 << int(ceil(log(time / deltaT) / log(2.0)));
	printf("Time: %.2f; Steps: %d\n", time, numberOfTimesteps);


	// re-sample 4 times during simulation
	const int sample_reset = numberOfTimesteps / 4;
	int nsample = 0;

	int numberOfInflowParticlesEachStep = cellDimensions.y * cellDimensions.z * meanParticlePerCell;

	int currentNumberOfParticles = 0;

	const int numberOfCells = cellDimensions.x * cellDimensions.y * cellDimensions.z;
	pnum = density * numberOfCells / float(meanParticlePerCell);

	cell* cellSamples = (cell*)malloc(numberOfCells * sizeof(cell));
	

	collisionInfo* collisionData = (collisionInfo*)malloc(numberOfCells * sizeof(collisionInfo));
	initializeCollision(collisionData, numberOfCells, vtemp);

	hiprandState_t* deviceRandomInflowStates = NULL;
	
	hipError_t cudaStatus = initializeCuda(&deviceRandomInflowStates, numberOfInflowParticlesEachStep);
	if (cudaStatus != hipSuccess) {
		printf("init kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
	}

	particle *allParticles = (particle*)malloc(numberOfInflowParticlesEachStep * sizeof(particle));
	particle *inflowParticleList = (particle*)malloc(numberOfInflowParticlesEachStep * sizeof(particle));

	clock_t clockTime;

	for (int t = 0; t < numberOfTimesteps; t++) {

		cudaStatus = inflowPotentialParticles(deviceRandomInflowStates, inflowParticleList, cellDimensions, meanParticlePerCell, vmean, vtemp);
		if (cudaStatus != hipSuccess) {
			printf("inflow failed: %s\n", hipGetErrorString(cudaStatus));
			return 1;
		}
		// Combine new particles with existing
		particle* newTotal = (particle*)malloc((numberOfInflowParticlesEachStep + currentNumberOfParticles) * sizeof(particle));
		memcpy(newTotal, inflowParticleList, numberOfInflowParticlesEachStep * sizeof(particle));
		if (currentNumberOfParticles != 0) {
			memcpy(newTotal + numberOfInflowParticlesEachStep, allParticles, currentNumberOfParticles * sizeof(particle));
		}
		free(allParticles);
		allParticles = newTotal;
		currentNumberOfParticles += numberOfInflowParticlesEachStep;

		int numToDelete = 0;
		moveAndIndexParticles(allParticles, currentNumberOfParticles, deltaT, cellDimensions, dividedCellDimensions, &numToDelete);

		// Clean up list of particles out of bounds and recompute cell map
		particle* cleanedParticles = removeParticlesOutofBounds(allParticles, currentNumberOfParticles, currentNumberOfParticles - numToDelete);
		currentNumberOfParticles -= numToDelete;
		free(allParticles);
		allParticles = cleanedParticles;

		if (t % sample_reset == 0)
		{
			clearCellInformation(cellSamples, numberOfCells);
			nsample = 0;
		}
		nsample++;
		
		sampleCellInformation(allParticles, currentNumberOfParticles, cellSamples, numberOfCells);

		clockTime = clock();

		collideParticles(allParticles,
			currentNumberOfParticles,
			collisionData,
			cellSamples,
			numberOfCells,
			nsample,
			numberOfCells,
			deltaT,
			cudaRandomHostGenerator);

		clockTime = clock() - clockTime;
		printf("[%d] time: %f to collide ", t, ((double)clockTime) / CLOCKS_PER_SEC);
		printf("num particles: %d\n", currentNumberOfParticles);
	}

	writeParticles(0, allParticles, currentNumberOfParticles);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	printf("Complete");

    return 0;
}

/* ============================== INITIALIZE =============================== */

__global__ void initRandomStates(unsigned int seed, hiprandState_t* states) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	/* we have to initialize the state */
	hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
		idx, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&states[idx]);
}

hipError_t initializeCuda(hiprandState_t **randomInflowStates, int numOfStates)
{
	int blockSize = numOfStates / NUM_OF_BLOCKS;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t hipError_t = hipSetDevice(0);

	hiprandState_t *dev_states;

	hipError_t = hipMalloc((void**) &dev_states, numOfStates * sizeof(hiprandState_t));
	if (hipError_t != hipSuccess) {
		printf("init malloc failed: %s\n", hipGetErrorString(hipError_t));
		return hipError_t;
	}

	initRandomStates <<<NUM_OF_BLOCKS, blockSize >>>(2, dev_states);
	
	*randomInflowStates = dev_states;

	return hipGetLastError();
}

/* ================================ HELPERS ================================ */

/*
	Compute a unit vector with a random orientation and uniform distribution
*/
__device__ void randomDirection(hiprandState_t seed, vect3d* vel)
{
	vel[0].x = 2.0 * hiprand_uniform(&seed) - 1;
	double A = sqrt(1. - vel[0].x * vel[0].x);
	double theta = hiprand_uniform(&seed) * 2 * HIP_PI_F;
	vel[0].y = A * cos(theta);
	vel[0].z = A * sin(theta);
}

/* =============================== 1. INFLOW =============================== */

__global__ void inflowKernel(int numOfParticles, hiprandState_t *randState, particle *particles, int dimX, int dimY, int dimZ, float vmean, float vtemp)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= numOfParticles) {
		return;
	}

	int k = idx % (dimX* dimY);
	int cellY = k % int(dimY);
	int cellZ = int(floorf(float(k) / float(dimY)));

	double dx = 2. / float(dimX);
	double dy = 2. / float(dimY);
	double dz = 2. / float(dimZ);

	double cx = -1 - dx;
	double cy = -1 + float(cellY) * dy;
	double cz = -1 + float(cellZ) * dz;

	particles[idx].position.x = cx + hiprand_uniform(&randState[idx]) * dx;
	particles[idx].position.y = cy + hiprand_uniform(&randState[idx]) * dy;
	particles[idx].position.z = cz + hiprand_uniform(&randState[idx]) * dz;

	randomDirection(randState[idx], &particles[idx].velocity);

	double rndVel = sqrt(-log(fmax(double(sqrt(hiprand_uniform(&randState[idx]))), 1e-200))) * vtemp;

	particles[idx].velocity.x = (particles[idx].velocity.x * rndVel) + vmean;
	particles[idx].velocity.y = particles[idx].velocity.y * rndVel;
	particles[idx].velocity.z = particles[idx].velocity.z * rndVel;

	particles[idx].index = 1;
	particles[idx].status = 0;
}

/* 
 Fill an array with new random particles to be exeucuted on

 Notes:
	Do I even need dev_a? Can I just use particle list?
 
 TODO:
	figure out how to build random seeds in seperate block
	Stress test number of particles

 */
hipError_t inflowPotentialParticles(hiprandState_t* deviceRandomStates, particle *particleList, vect3d cellDimensions, int meanParticlePerCell, float vmean, float vtemp) {
	
	int numOfPoints = cellDimensions.y * cellDimensions.z * meanParticlePerCell;

	int numberOfBlocks = ceil(double(numOfPoints) / double(MAX_THREAD_PER_BLOCK));

	int size = numOfPoints * sizeof(particle);

	particle *dev_a;
	hipError_t cudaStatus = hipMalloc((void**)&dev_a, size);
	if (cudaStatus != hipSuccess) {
		printf("inflow malloc failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	inflowKernel <<<numberOfBlocks, MAX_THREAD_PER_BLOCK >>>(numOfPoints, deviceRandomStates, dev_a, cellDimensions.x, cellDimensions.y, cellDimensions.z, vmean, vtemp);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("inflow launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(particleList, dev_a, size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("inflow memcpy failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	return hipFree(dev_a);
}



/* =========================== 2. MOVE PARTICLES =========================== */

/*
	TODO:
		MOVE BRANCHIGN COLLISION LOGIC OUTSIDE OF KERNEL
*/
__global__ void moveParticlesKernel(int* deviceDeletionCount, particle* particles, int numParticles, float deltaTime, int dimX, int dimY, int dimZ, float divX, float divY, float divZ) {
	
	extern __shared__ int sdata[];
	sdata[threadIdx.x] = 0;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= numParticles) {
		return;
	}

	float newPosX = particles[idx].position.x + (particles[idx].velocity.x * deltaTime);
	float newPosY = particles[idx].position.y + (particles[idx].velocity.y * deltaTime);
	float newPosZ = particles[idx].position.z + (particles[idx].velocity.z * deltaTime);


	// Where did it actually pass through the plate..
	double t = (particles[idx].position.x - PLATE_X) / (particles[idx].position.x - newPosX);
	float pointOfCollisionY = (particles[idx].position.y * (1.0 - t)) + (newPosY * t);
	float pointOfCollisionZ = (particles[idx].position.z * (1.0 - t)) + (newPosZ * t);

	// Actually collided..
	if (((particles[idx].position.x < PLATE_X && newPosX > PLATE_X) ||
		(particles[idx].position.x > PLATE_X && newPosX < PLATE_X)) && (pointOfCollisionY < PLATE_DY && pointOfCollisionY > -PLATE_DY) &&
		(pointOfCollisionZ < PLATE_DZ && pointOfCollisionZ > -PLATE_DZ))
	{
		newPosX = newPosX - 2 * (newPosX - PLATE_X);
		particles[idx].velocity.x = -particles[idx].velocity.x;
		particles[idx].status = 1;
	}


	newPosY = newPosY + (2.0 * ((newPosY < -1) - (newPosY > 1)));
	newPosZ = newPosZ + (2.0 * ((newPosZ < -1) - (newPosZ > 1)));

	// Assign particle positions and index
	particles[idx].position.x = newPosX;
	particles[idx].position.y = newPosY;
	particles[idx].position.z = newPosZ;

	int i = int(fmin(floor((newPosX + 1.0) / divX), double(dimX - 1)));
	int j = int(fmin(floor((newPosY + 1.0) / divY), double(dimY - 1)));
	int k = int(fmin(floor((newPosZ + 1.0) / divZ), double(dimZ - 1)));
	particles[idx].index = i * dimY * dimZ + j * dimZ + k;

	int deletion = 0;
	if (newPosX > 1.0 || newPosX < -1.0) {
		deletion = 1;
		particles[idx].status = -1;
	}

	sdata[threadIdx.x] = deletion;
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
		if (threadIdx.x < s) {
			sdata[threadIdx.x] += sdata[threadIdx.x + s];
		}
		__syncthreads();
	}
	if (threadIdx.x == 0) {
		deviceDeletionCount[blockIdx.x] = sdata[0];
	}
}

/*
	Move particles appropriately and marks those out of bounds with a flag for deletion.
	Reindexes particles not marked for deletion

	TODO:
		Some how parrallel sum how many particles now need to be deleted..
*/
hipError_t moveAndIndexParticles(particle* particleList, int numOfParticles, float deltaTime, vect3d cellDimensions, vect3d dividedCellDimensions, int* numToDelete) {
	int numberOfBlocks = ceil(double(numOfParticles) / double(MAX_THREAD_PER_BLOCK));
	
	particle *dev_a;

	int *deviceDeletionCount;

	hipError_t cudaStatus = hipMalloc((void**)&dev_a, numOfParticles*sizeof(particle));
	if (cudaStatus != hipSuccess) {
		printf("move&index malloc failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&deviceDeletionCount, numberOfBlocks * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("move&index malloc failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(dev_a, particleList, numOfParticles * sizeof(particle), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("move&index memcpy to device failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	moveParticlesKernel <<<numberOfBlocks, MAX_THREAD_PER_BLOCK, MAX_THREAD_PER_BLOCK * sizeof(int) >>>(deviceDeletionCount, dev_a, numOfParticles, deltaTime, cellDimensions.x, cellDimensions.y, cellDimensions.z, dividedCellDimensions.x, dividedCellDimensions.y, dividedCellDimensions.z);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("move&index launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}


	cudaStatus = hipMemcpy(particleList, dev_a, numOfParticles * sizeof(particle), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("move&index memcpy to host failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	int* deletionCount = (int*)malloc(numberOfBlocks * sizeof(int));
	cudaStatus = hipMemcpy(deletionCount, deviceDeletionCount, numberOfBlocks * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("move&index memcpy to host failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	for (int i = 0; i < numberOfBlocks; i++) {
		*numToDelete += deletionCount[i];
	}
	free(deletionCount);

	cudaStatus = hipFree(deviceDeletionCount);
	if (cudaStatus != hipSuccess) {
		printf("move&index free count failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	return hipFree(dev_a);
}

/* ========================== 3. REMOVE PARTICLES ========================== */

particle* removeParticlesOutofBounds(particle* particles, int originalParticleSize, int newSize) {
	particle* newParticleList = (particle*)malloc(newSize * sizeof(particle));
	int added = 0;
	for (int i = 0; i < originalParticleSize; i++) {
		if (particles[i].status != -1) {
			newParticleList[added] = particle(particles[i]);
			added += 1;
		}
	}
	return newParticleList;
}

/* ============================== 4. SAMPLING ============================== */

__global__ void clearCellsKernel(cell* cells, int numCells) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= numCells) {
		return;
	}
	cells[idx].numberOfParticles = 0;
	cells[idx].velocity.x = 0;
	cells[idx].velocity.y = 0;
	cells[idx].velocity.z = 0;
	cells[idx].energy = 0;
}

/*
	TODO:
		CHECK IF THIS IS ANY BETTER THAN DOING IT LINEARLY
*/
hipError_t	clearCellInformation(cell* cells, int numCells) {
	int numberOfBlocks = ceil(double(numCells) / double(MAX_THREAD_PER_BLOCK));
	cell *deviceCells;

	hipError_t cudaStatus = hipMalloc((void**)&deviceCells, numCells * sizeof(cell));
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}

	hipMemcpy(deviceCells, cells, numCells * sizeof(cell), hipMemcpyHostToDevice);

	clearCellsKernel <<<numberOfBlocks, MAX_THREAD_PER_BLOCK >>>(deviceCells, numCells);

	cudaStatus = hipMemcpy(cells, deviceCells, numCells * sizeof(cell), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}

	hipFree(deviceCells);

	return cudaStatus;
}

__global__ void sampleCellsKernel(cell* cells, int cellSize, particle* particles, int particleSize, particle*** cellToParticleMap) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	for (int particleIndex = 0; particleIndex < particleSize; particleIndex++)
	{
		if (particles[particleIndex].index == idx)
		{
			cells[idx].numberOfParticles = cells[idx].numberOfParticles + 1;
			cells[idx].velocity.x = particles[particleIndex].velocity.x + cells[idx].velocity.x;
			cells[idx].velocity.y = particles[particleIndex].velocity.y + cells[idx].velocity.y;
			cells[idx].velocity.z = particles[particleIndex].velocity.z + cells[idx].velocity.z;
			cells[idx].energy = cells[idx].energy + (
				.5 * 
				((particles[particleIndex].velocity.x * particles[particleIndex].velocity.x) +
				(particles[particleIndex].velocity.y * particles[particleIndex].velocity.y) +
				(particles[particleIndex].velocity.z * particles[particleIndex].velocity.z)));
		}
	}
}

void sampleCellInformation(particle* particles, int numOfParticles, cell* cells, int numCells) {
	
	for (int cellIndex = 0; cellIndex < numCells; cellIndex++) {
		cells[cellIndex].currentNumberOfParticles = 0;
	}

	for (int p = 0; p < numOfParticles; p++) {
		int cellIndex = particles[p].index;
		cells[cellIndex].currentNumberOfParticles++;
		cells[cellIndex].numberOfParticles++;
		cells[cellIndex].velocity += particles[p].velocity;
		cells[cellIndex].energy += .5* dot(particles[p].velocity, particles[p].velocity);
	}

}


/* ============================ Cell Collision ============================= */

__global__ void preCollisionKernel(cell* cells, collisionInfo* collisionData, int cellSize, float nSample, float deltaT, float cellVol, float moleculesPerParticle, int *numOfRandomNumbers) {

	extern __shared__ int sdata[];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[threadIdx.x] = 0;

	if (idx >= cellSize) {
		return;
	}

	// Compute mean and instantaneous particle numbers for the cell
	float n_mean = float(cells[idx].numberOfParticles) / float(nSample);
	float n_instant = cells[idx].currentNumberOfParticles;

	// Compute a number of particles that need to be selected for collision tests
	float select = n_instant * n_mean * moleculesPerParticle * collisionData[idx].maxCollisionRate * deltaT / cellVol + collisionData[idx].collisionRemainder;
	
	// We can only check an integer number of collisions in any timestep
	collisionData[idx].nSelect = int(select);

	// The remainder collision fraction is saved for next timestep
	collisionData[idx].collisionRemainder = select - float(collisionData[idx].nSelect);

	sdata[threadIdx.x] = collisionData[idx].nSelect * (n_instant >= 2) * 5;
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
		if (threadIdx.x < s) {
			sdata[threadIdx.x] += sdata[threadIdx.x + s];
		}
		__syncthreads();
	}
	if (threadIdx.x == 0) {
		numOfRandomNumbers[blockIdx.x] = sdata[0];
	}

}


// Computes a unit vector with a random orientation and uniform distribution
inline vect3d randomDir(float rndOne, float rndTwo)
{
	double B = 2. * rndOne - 1;
	double A = sqrt(1. - B * B);
	double theta = rndTwo * 2 * HIP_PI_F;
	return vect3d(B, A * cos(theta), A * sin(theta));
}

void collideParticles(
	particle* particleList,
	int particleListSize,
	collisionInfo* collisionData,
	cell* cellData,
	int cellDataSize,
	int nsample, 
	float cellvol, 
	float deltaT,
	hiprandGenerator_t cudaRandomHostGenerator)
{

	// Compute number of particles per cell and compute a set of pointers
	// from each cell to the corresponding particles
	int* np = (int*)malloc(cellDataSize * sizeof(int));
	int* cnt = (int*)malloc(cellDataSize * sizeof(int));
	for (int i = 0; i < cellDataSize; ++i)
	{
		np[i] = 0;
		cnt[i] = 0;
	}

	for (int p = 0; p < particleListSize; p++)
	{
		np[particleList[p].index]++;
	}

	// Offsets will contain the index in the pmap data structure where
	// the pointers to particles for the given cell will begin
	int* offsets = (int*)malloc((cellDataSize + 1) * sizeof(int));
	offsets[0] = 0;
	for (int i = 0; i < cellDataSize; ++i)
	{
		offsets[i + 1] = offsets[i] + np[i];
	}

	// pmap is a structure of pointers from cells to particles, note
	// since there may be many particles per cell, the offsets need to
	// be used to access particles from this data structure.
	particle** pmap = (particle**)malloc(offsets[cellDataSize] * sizeof(particle*));
	for (int p = 0; p < particleListSize; p++)
	{
		int i = particleList[p].index;
		pmap[cnt[i] + offsets[i]] = &(particleList[p]);
		cnt[i]++;
	}

	free(cnt);
	
	cell* deviceCellData;
	hipError_t cudaStatus = hipMalloc((void**)&deviceCellData, cellDataSize * sizeof(cell));
	if (cudaStatus != hipSuccess) {
		printf("Pcol malloc failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}
	
	collisionInfo* deviceCollisionData;
	cudaStatus = hipMalloc((void**)&deviceCollisionData, cellDataSize * sizeof(collisionInfo));
	if (cudaStatus != hipSuccess) {
		printf("Pcol malloc failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	int* deviceRandomNumbersNeeded;
	cudaStatus = hipMalloc((void**)&deviceRandomNumbersNeeded, cellDataSize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("Pcol malloc failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	cudaStatus = hipMemcpy(deviceCellData, cellData, cellDataSize * sizeof(cell), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("Pcol memcpy to device failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	cudaStatus = hipMemcpy(deviceCollisionData, collisionData, cellDataSize * sizeof(collisionInfo), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("Pcol memcpy to device failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	int numberOfBlocks = ceil(double(cellDataSize) / double(MAX_THREAD_PER_BLOCK));

	preCollisionKernel<<<numberOfBlocks , MAX_THREAD_PER_BLOCK, MAX_THREAD_PER_BLOCK * sizeof(int)>>>(deviceCellData, deviceCollisionData, cellDataSize, nsample, deltaT, cellvol, pnum, deviceRandomNumbersNeeded);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("Pcol launch failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	int* blockAggRandomNumbersNeeded = (int*)malloc(numberOfBlocks * sizeof(int));
	cudaStatus = hipMemcpy(blockAggRandomNumbersNeeded, deviceRandomNumbersNeeded, numberOfBlocks * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("Pcol memcpy to host failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	int randomNumbersNeeded = 0;
	for (int i = 0; i < numberOfBlocks; i++) {
		randomNumbersNeeded += blockAggRandomNumbersNeeded[i];
	}
	free(blockAggRandomNumbersNeeded);

	float* randomNumbers = (float *)malloc(randomNumbersNeeded * sizeof(float));
	float* deviceRandomNumbers;
	cudaStatus = hipMalloc((void **)&deviceRandomNumbers, randomNumbersNeeded * sizeof(float));
	
	hiprandGenerateUniform(cudaRandomHostGenerator, deviceRandomNumbers, randomNumbersNeeded);
	cudaStatus = hipMemcpy(randomNumbers, deviceRandomNumbers, randomNumbersNeeded * sizeof(float), hipMemcpyDeviceToHost);
	int randomNumbersUsed = 0;

	cudaStatus = hipMemcpy(collisionData, deviceCollisionData, cellDataSize * sizeof(collisionInfo), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("Pcol memcpy to host failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	// Loop over cells and select particles to perform collisions
	for (int cellIndex = 0; cellIndex < cellDataSize; ++cellIndex)
	{
		if (collisionData[cellIndex].nSelect > 0)
		{ // selected particles for collision
			if (cellData[cellIndex].currentNumberOfParticles < 2)
			{ // if not enough particles for collision, wait until
			  // we have enough
				collisionData[cellIndex].collisionRemainder += collisionData[cellIndex].nSelect;
			}
			else
			{
				// Select nselect particles for possible collision
				float cmax = collisionData[cellIndex].maxCollisionRate;
				for (int c = 0; c < collisionData[cellIndex].nSelect; ++c)
				{

					// select two points in the cell
					int pt1 = min(int(floor(randomNumbers[randomNumbersUsed] * cellData[cellIndex].currentNumberOfParticles)), cellData[cellIndex].currentNumberOfParticles - 1);
					int pt2 = min(int(floor(randomNumbers[randomNumbersUsed + 1] * cellData[cellIndex].currentNumberOfParticles)), cellData[cellIndex].currentNumberOfParticles - 1);

					// Make sure they are unique points
					while (pt1 == pt2) {
						pt2 = min(int(floor(ranf() * cellData[cellIndex].currentNumberOfParticles)), cellData[cellIndex].currentNumberOfParticles - 1);
					}
					// Compute the relative velocity of two particles
					vect3d v1 = pmap[offsets[cellIndex] + pt1]->velocity;
					vect3d v2 = pmap[offsets[cellIndex] + pt2]->velocity;
					vect3d vr = v1 - v2;
					float vrm = norm(vr);

					// Compute collision  rate for hard sphere model
					float crate = sigmak * vrm;
					if (crate > cmax) {
						cmax = crate;
					}
					
					// Check if these particles actually collide
					if (randomNumbers[randomNumbersUsed + 2] < crate / collisionData[cellIndex].maxCollisionRate)
					{

						// Collision Accepted, adjust particle velocities
						// Compute center of mass velocity, vcm
						vect3d vcm = .5 * (v1 + v2);
						// Compute random perturbation that conserves momentum
						vect3d vp = randomDir(randomNumbers[randomNumbersUsed + 3], randomNumbers[randomNumbersUsed + 4]) * vrm;

						// Adjust particle velocities to reflect collision
						pmap[offsets[cellIndex] + pt1]->velocity = vcm + 0.5 * vp;
						pmap[offsets[cellIndex] + pt2]->velocity = vcm - 0.5 * vp;

						pmap[offsets[cellIndex] + pt1]->status = 2;
						pmap[offsets[cellIndex] + pt2]->status = 2;
					}
					randomNumbersUsed += 5;
				}
				// Update the maximum collision rate to be used in future timesteps
				// for determining number of particles to select.
				collisionData[cellIndex].maxCollisionRate = cmax;
			}
		}
	
	}

	free(pmap);
	free(np);
	free(offsets);
	free(randomNumbers);

	hipFree(deviceRandomNumbers);

	cudaStatus = hipFree(deviceCellData);
	if (cudaStatus != hipSuccess) {
		printf("Pcol free cell failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

	cudaStatus = hipFree(deviceCollisionData);
	if (cudaStatus != hipSuccess) {
		printf("Pcol free cell failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}

}